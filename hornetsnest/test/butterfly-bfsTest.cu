/*
 * Copyright (c) 2020, NVIDIA CORPORATION
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 * 
 * 1. Redistributions of source code must retain the above copyright notice,
 *    this list of conditions and the following disclaimer.
 * 
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 * 
 * 3. Neither the name of the copyright holder nor the names of its contributors
 *    may be used to endorse or promote products derived from this software without
 *    specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
 * ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include <vector>

#include <omp.h>
// #include <time.h> 
#include <sys/time.h>

#include "Static/butterfly/butterfly-bfs.cuh"
#include "Static/butterfly/butterfly-bfsOperators.cuh"

using namespace std;
#include <array>

using namespace graph;
using namespace graph::structure_prop;
using namespace graph::parsing_prop;

#define CHECK_ERROR(str) \
    {hipError_t err; err = hipGetLastError(); if(err!=0) {printf("ERROR %s:  %d %s\n", str, err, hipGetErrorString(err)); fflush(stdout); exit(0);}}

using namespace timer;
using namespace hornets_nest;

// A recursive binary search function for partitioning the vertices.
// Vertices are NOT split amongst the cores\GPUs thus
// we returns the vertex id with the smallest value larger than x (which is the edge partition)

template <typename t,typename pos_t>
pos_t vertexBinarySearch(const t *offsets, pos_t l, pos_t r, t x) 
{ 
    if (r >= l) { 
        pos_t mid = l + (r - l) / 2L; 
  
        // If the element is present at the middle itself 
        if (offsets[mid] == x) // perfect load balancing
            return mid; 
  
        // Check left subarray
        if (offsets[mid] > x) 
            return vertexBinarySearch(offsets, l, mid - 1L, x); 
        else
        // Check right subarray 
            return vertexBinarySearch(offsets, mid + 1L, r, x); 
    } 
  
    // Return the vertex id of the smallest vertex with an offset greater than x. 
    return l; 
} 

#include <vector>
#include <algorithm>
using vecPair = pair<int,int>;
vector< vecPair > vecInput;

int main(int argc, char* argv[]) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    using namespace graph;

    vert_t *h_cooSrc,*h_cooDst;
    int64_t nV,nE;

    int64_t numGPUs=4; int64_t logNumGPUs=2; int64_t fanout=1;
    int64_t minGPUs=1,maxGPUs=16;
    // bool isLrb=false;
    int isLrb=0,onlyLrb=0,onlyFanout4=0;

    vert_t startRoot = 0;//(vert_t)graph.max_out_degree_id();
    vert_t root = startRoot;

    if (argc>=3){
        minGPUs = atoi(argv[2]);
    }
    if (argc>=4){
        maxGPUs = atoi(argv[3]);
    }

    if (argc>=5){
        onlyLrb = atoi(argv[4]);
    }
    if (argc>=6){
        onlyFanout4 = atoi(argv[5]);
    }

    hipSetDevice(0);
    // if(0)
    // {
    //     ParsingProp pp(graph::detail::ParsingEnum::NONE);
    //     // graph::GraphStd<int64_t, int64_t> graph(UNDIRECTED);
    //     graph::GraphStd<int64_t, int64_t> graph(DIRECTED);
    //     // graph::GraphStd<vert_t, eoff_t> graph(DIRECTED);
    //     graph.read(argv[1],pp,true);

    //     auto cooGraph = graph.coo_ptr();

    //     h_cooSrc = new vert_t[2*graph.nE()];
    //     h_cooDst = new vert_t[2*graph.nE()];

    //     #pragma omp parallel for 
    //     for(int64_t i=0; i < graph.nE(); i++){
    //         // if(i>(graph.nE()-50))
    //         //     printf("%ld %ld\n",cooGraph[i].first,cooGraph[i].second);
    //         h_cooSrc[i] = cooGraph[i].first;
    //         h_cooDst[i] = cooGraph[i].second;
    //         h_cooSrc[i+graph.nE()] = cooGraph[i].second;
    //         h_cooDst[i+graph.nE()] = cooGraph[i].first;
    //     }
    //     nV = graph.nV();
    //     nE = 2*graph.nE();

    //     printf("Number of vertices is : %ld\n", nV);
    //     printf("Number of edges is    : %ld\n", nE);


    //     if(reOrgFlag){
    //         printf("REORDERING!!\n");
    //         for(int mul= 1; mul < 10; mul++)
    //         {
    //             int m = 1 << mul;
    //             auto nVdivM = nV/m;
    //             #pragma omp parallel for
    //             for(int64_t i=0; i < nE; i++){
    //                 if((h_cooSrc[i]%m)==0){
    //                     if((h_cooSrc[i]+nVdivM)>=nV){
    //                         h_cooSrc[i] = h_cooSrc[i]%nVdivM;
    //                     }else{
    //                         h_cooSrc[i]+=nVdivM;
    //                     }
    //                 }

    //                 if((h_cooDst[i]%m)==0){
    //                     if((h_cooDst[i]+nVdivM)>=nV){
    //                         h_cooDst[i] = h_cooDst[i]%nVdivM;
    //                     }else{
    //                         h_cooDst[i]+=nVdivM;
    //                     }
    //                 }
    //             }
    //         }
    //     }
    // }
    // else{
        ParsingProp pp(graph::detail::ParsingEnum::NONE);
        graph::GraphStd<int64_t, int64_t> graph(UNDIRECTED);
        graph.read(argv[1],pp,true);

        nV = graph.nV();
        nE = graph.nE();


        h_cooDst=nullptr;
        h_cooSrc=nullptr;
 
        // printf("Number of vertices is : %ld\n", nV);
        // printf("Number of edges is    : %ld\n", nE);
        // fflush(stdout);

    omp_set_num_threads(maxGPUs);
    // hornets_nest::gpu::initializeRMMPoolAllocation(0,maxGPUs);//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.


    hipSetDevice(0);
    
    #pragma omp parallel
    {      
        int64_t thread_id = omp_get_thread_num ();
        hipSetDevice(thread_id);

        for(int64_t g=0; g<maxGPUs; g++){
            if(g!=thread_id){
                int isCapable;
                hipDeviceCanAccessPeer(&isCapable,thread_id,g);
                if(isCapable==1){
                    hipDeviceEnablePeerAccess(g,0);
                }
            }
        }
    }

    hipSetDevice(0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int64_t edgeSplits [maxGPUs+1];
    int fanoutArray[2]={1,4};

    for(int g=minGPUs; g<=maxGPUs;g++){
        hipSetDevice(0);
        numGPUs=g;

        int logNumGPUsArray[17] = {0,1,2,2,2,3,3,3,3,4,4,4,4,4,4,4,4};
        logNumGPUs = logNumGPUsArray[numGPUs];

        omp_set_num_threads(numGPUs);


        using vertPtr = vert_t*;

        vert_t* localOffset=nullptr;
        vert_t* edges=nullptr;

        hipSetDevice(0);


        using HornetGraphPtr = HornetGraph*;

        HornetGraphPtr hornetArray[numGPUs];
        vert_t maxArrayDegree[numGPUs];
        vert_t maxArrayId[numGPUs];


        #pragma omp parallel
        {      
            int64_t thread_id = omp_get_thread_num ();
            hipSetDevice(thread_id);

            int64_t upperNV = nV;
            if(upperNV%numGPUs){
                upperNV = nV - (nV%numGPUs) + numGPUs;
            }
            int64_t upperNE = nE;
            if(upperNE%numGPUs){
                upperNE = nE - (nE%numGPUs) + numGPUs;
            }

            int64_t edgeVal = ((thread_id+1L) * upperNE) /numGPUs ;
            if (edgeVal>nE)
                edgeVal = nE;
            int64_t zero=0;
            edgeSplits[thread_id+1] = vertexBinarySearch(graph.csr_out_offsets(),zero, nV+1L, (edgeVal));

            if(thread_id == 0 )
                edgeSplits[0]=0;
            #pragma omp barrier

            int64_t my_start,my_end, my_edges;

            my_start = edgeSplits[thread_id];
            my_end  = edgeSplits[thread_id+1];
            my_edges = graph.csr_out_offsets()[my_end]-graph.csr_out_offsets()[my_start];

            localOffset = (vert_t*)malloc(sizeof(vert_t)*(nV+1));
            edges       = (vert_t*)malloc(sizeof(vert_t)*(my_edges));

            int64_t i=0;
            for(int64_t u=my_start; u<my_end; u++){
                int64_t d_size=graph.csr_out_offsets()[u+1]-graph.csr_out_offsets()[u];
                for (int64_t d=0; d<d_size; d++){
                    edges[i++]=(vert_t) graph.csr_out_edges()[(graph.csr_out_offsets()[u]+d)];
                }
            }

            // printf("%ld %ld %ld %ld %ld %ld\n", thread_id,my_start,my_end, my_edges,graph.csr_out_offsets()[my_start],graph.csr_out_offsets()[my_end]);
            // fflush(stdout);

            for(int64_t v=0; v<(nV+1); v++){
                localOffset[v]=0;
            }
            for(int64_t v=(my_start); v<nV; v++){
                localOffset[v+1] = localOffset[v] + (graph.csr_out_offsets()[v+1]-graph.csr_out_offsets()[v]);
            }

            HornetInit hornet_init(nV,my_edges, localOffset,edges);

            hornetArray[thread_id] = new HornetGraph(hornet_init);

            #pragma omp barrier
            maxArrayDegree[thread_id]   = hornetArray[thread_id]->max_degree();
            maxArrayId[thread_id]       = hornetArray[thread_id]->max_degree_id();
        }

        vert_t max_d    = maxArrayDegree[0];
        vert_t max_id   = maxArrayId[0];
        for(int m=1;m<numGPUs; m++){
            if(max_d<maxArrayDegree[m]){
                max_d   = maxArrayDegree[m];
                max_id  = maxArrayId[m];
            }
        }
        omp_set_num_threads(numGPUs);


        for(int f=0; f<2 ; f++){
            if(f==0 && onlyFanout4)
                continue;
            fanout=fanoutArray[f];

            vert_t my_start_array[maxGPUs],my_end_array[maxGPUs];
            for(int thread_id=0; thread_id<numGPUs; thread_id++){
                my_start_array[thread_id]  = edgeSplits[thread_id];
                my_end_array[thread_id]    = edgeSplits[thread_id+1];
            }

            multiButterfly mBF(hornetArray,numGPUs,fanout);
            mBF.setVertexBoundries(my_start_array,my_end_array);

            hipSetDevice(0);

            for(int lrb=0; lrb<2; lrb++){
                if(lrb==0 && onlyLrb)
                    continue;
                isLrb=lrb;

                printf("%s,",argv[1]);
                printf("%ld,%ld,",nV,nE);
                printf("%ld,",numGPUs);
                printf("%ld,",logNumGPUs);
                printf("%ld,",fanout);
                printf("%d,",isLrb);
                printf("%d,",max_id); // Starting root

                double totalTime = 0;
                int totatLevels = 0;
                root=max_id;
                int totalRoots = 100;
                double timePerRoot[totalRoots];
                for(int64_t i=0; i<totalRoots; i++){
                    if(i>0){
                        root++;
                        if(root>nV)
                            root=0;
                    }

                    hipSetDevice(0);

                    hipEventRecord(start); 
                    hipEventSynchronize(start); 

                    mBF.reset();
                    mBF.setRootandQueue(root);
                    mBF.run();

                    hipEventRecord(stop);
                    hipEventSynchronize(stop);
                    float milliseconds = 0;
                    hipEventElapsedTime(&milliseconds, start, stop);  
                    // printf("%f,", milliseconds/1000.0);
                    timePerRoot[i] = milliseconds/1000.0;
                    // std::cout << "Number of levels is : " << front << std::endl;
                    // std::cout << "The number of traversed vertices is : " << countTraversed << std::endl;

                    // totatLevels +=front;
                    totatLevels += mBF.front;

                }

                std::sort(timePerRoot,timePerRoot+totalRoots);
                int filterRoots = totalRoots/2;
                for(int root = 0; root < filterRoots; root++){
                    totalTime += timePerRoot[filterRoots+totalRoots/4];
                }
                printf("%lf,", totalTime);
                printf("%lf,", totalTime/(double)filterRoots);
                printf("%d,",  filterRoots);
                printf("%d,", totatLevels);

                printf("\n");

            }
        }


        // #pragma omp parallel
        for(int i=0; i< numGPUs; i++) // very weird compiler error.
        {      
            // int64_t thread_id = omp_get_thread_num ();
            int64_t thread_id = i;
            hipSetDevice(thread_id);

            delete hornetArray[thread_id];
        }

        hipSetDevice(0);

        if(localOffset!=nullptr)
            delete[] localOffset; 
        if(edges!=nullptr)
            delete[] edges;

    }

    hipSetDevice(0);
    omp_set_num_threads(maxGPUs);

    #pragma omp parallel
    {      
        int64_t thread_id = omp_get_thread_num ();
        hipSetDevice(thread_id);

        for(int64_t g=0; g<numGPUs; g++){
            if(g!=thread_id){
                int isCapable;
                hipDeviceCanAccessPeer(&isCapable,thread_id,g);
                if(isCapable==1){
                    hipDeviceDisablePeerAccess(g);
                }
            }
        }
    }

    if(h_cooSrc!=nullptr)
        delete[] h_cooSrc;
    if(h_cooDst!=nullptr)
        delete[] h_cooDst;


    // hornets_nest::gpu::finalizeRMMPoolAllocation(maxGPUs);

    return 0;


}


