#include "hip/hip_runtime.h"

#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include <vector>

#include <omp.h>

#include "Static/butterfly/butterfly-bfs.cuh"
#include "Static/butterfly/butterfly-bfsOperators.cuh"

using namespace std;
#include <array>

using namespace graph;
using namespace graph::structure_prop;
using namespace graph::parsing_prop;

#define CHECK_ERROR(str) \
    {hipError_t err; err = hipGetLastError(); if(err!=0) {printf("ERROR %s:  %d %s\n", str, err, hipGetErrorString(err)); fflush(stdout); exit(0);}}

using namespace timer;
using namespace hornets_nest;



// A recursive binary search function for partitioning the vertices.
// Vertices are NOT split amongst the cores\GPUs thus
// we returns the vertex id with the smallest value larger than x (which is the edge partition)

template <typename t,typename pos_t>
pos_t vertexBinarySearch(const t *offsets, pos_t l, pos_t r, t x) 
{ 
    if (r >= l) { 
        pos_t mid = l + (r - l) / 2L; 
  
        // If the element is present at the middle itself 
        if (offsets[mid] == x) // perfect load balancing
            return mid; 
  
        // Check left subarray
        if (offsets[mid] > x) 
            return vertexBinarySearch(offsets, l, mid - 1L, x); 
        else
        // Check right subarray 
            return vertexBinarySearch(offsets, mid + 1L, r, x); 
    } 
  
    // Return the vertex id of the smallest vertex with an offset greater than x. 
    return l; 
} 


int main(int argc, char* argv[]) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;
    using namespace graph;

    // GraphStd<vert_t, vert_t> graph(UNDIRECTED);
    graph::GraphStd<int64_t,int64_t> graph(UNDIRECTED );
    graph.read(argv[1]);
    // graph.read(argv[1], RANDOMIZE);
    // CommandLineParam cmd(graph, argc, argv,false);
    Timer<DEVICE> TM;

    // int numGPUs=2; int logNumGPUs=1;
     // int numGPUs=4; int logNumGPUs=2;
    // int numGPUs=8; int logNumGPUs=3;
    // int numGPUs=16; int logNumGPUs=4;


    int64_t numGPUs=4; int64_t logNumGPUs=2; int64_t fanout=1;
    int64_t minGPUs=1,maxGPUs=16;
    // bool isLrb=false;
    int isLrb=0,onlyLrb=0,onlyFanout4=0;

    vert_t startRoot = (vert_t)graph.max_out_degree_id();
    vert_t root = startRoot;

    if (argc>=3){
        minGPUs = atoi(argv[2]);
    }
    if (argc>=4){
        maxGPUs = atoi(argv[3]);
    }

    if (argc>=5){
        onlyLrb = atoi(argv[4]);
    }
    if (argc>=6){
        onlyFanout4 = atoi(argv[5]);
    }

    // omp_set_num_threads(numGPUs);

    omp_set_num_threads(maxGPUs);

    #pragma omp parallel
    {      
        int64_t thread_id = omp_get_thread_num ();
        hipSetDevice(thread_id);

        for(int64_t g=0; g<maxGPUs; g++){
            if(g!=thread_id){
                int isCapable;
                hipDeviceCanAccessPeer(&isCapable,thread_id,g);
                if(isCapable==1){
                    hipDeviceEnablePeerAccess(g,0);
                }
            }
        }
    }

    hipSetDevice(0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    butterfly_communication bfComm[numGPUs];
    int64_t edgeSplits [16+1];
    int fanoutArray[2]={1,4};
    // std::array<int,2> fanoutArray{1,4};


    // for(int f=0; f<(int)fanoutArray.size() ; f++){
    for(int f=0; f<2 ; f++){
        if(f==0 && onlyFanout4)
            continue;
        fanout=fanoutArray[f];
    for(int lrb=0; lrb<2; lrb++){
        if(lrb==0 && onlyLrb)
            continue;
        isLrb=lrb;
    for(int g=minGPUs; g<=maxGPUs;g++){
        numGPUs=g;
        // int divg=numGPUs;

        int logNumGPUsArray[17] = {0,1,2,2,2,3,3,3,3,4,4,4,4,4,4,4,4};
        logNumGPUs = logNumGPUsArray[numGPUs];
        // logNumGPUs=0;
        // while(divg/2){
        //     logNumGPUs++;
        //     divg=divg/2;
        // }
        // if(divg>0 && numGPUs%)
        //     logNumGPUs++;

        // logNumGPUs = (int) log2(numGPUs); 
        // logNumGPUs = 31 -  __builtin_clz(numGPUs);
        omp_set_num_threads(numGPUs);


        root=startRoot;

        printf("%s,",argv[1]);
        printf("%ld,%ld,",graph.nV(),graph.nE());
        printf("%ld,",numGPUs);
        printf("%ld,",logNumGPUs);        
        printf("%ld,",fanout);
        printf("%d,",isLrb);
        printf("%d,",startRoot); // Starting root


            // printf("Waiting for enter\n");
            // int stam=0;
            // stam = scanf("%d",&stam);
            // printf("%d\n",stam+1);


        for(int64_t i=0; i<10; i++){

            if(i>0){
                root++;
                if(root>graph.nV())
                    root=0;
            }


            #pragma omp parallel
            {      
                int64_t thread_id = omp_get_thread_num ();
               // if(thread_id==0){
               //     printf(", %d ,",omp_get_num_threads() );
               // }
                hipSetDevice(thread_id);

                int64_t nV = graph.nV(); int64_t nE = graph.nE();

                int64_t upperNV = nV;
                if(upperNV%numGPUs){
                    upperNV = nV - (nV%numGPUs) + numGPUs;
                }
                int64_t upperNE = nE;
                if(upperNE%numGPUs){
                    upperNE = nE - (nE%numGPUs) + numGPUs;
                }

                int64_t edgeVal = ((thread_id+1L) * upperNE) /numGPUs ;
                if (edgeVal>nE)
                    edgeVal = nE;
                int64_t zero=0;
                edgeSplits[thread_id+1] = vertexBinarySearch(graph.csr_out_offsets(),zero, nV+1L, (edgeVal));
                // printf("%ld %ld %ld\n",thread_id,edgeSplits[thread_id+1],edgeVal);

                if(thread_id == 0 )
                    edgeSplits[0]=0;

                #pragma omp barrier

                int64_t my_start,my_end,my_edges;

                my_start = edgeSplits[thread_id];
                my_end  = edgeSplits[thread_id+1];
                my_edges = graph.csr_out_offsets()[my_end]-graph.csr_out_offsets()[my_start];

                vert_t* localOffset = (vert_t*)malloc(sizeof(vert_t)*(nV+1));
                vert_t* edges       = (vert_t*)malloc(sizeof(vert_t)*(my_edges));

                int64_t i=0;
                for(int64_t u=my_start; u<my_end; u++){
                    int64_t d_size=graph.csr_out_offsets()[u+1]-graph.csr_out_offsets()[u];
                    for (int64_t d=0; d<d_size; d++){
                        edges[i++]=(vert_t) graph.csr_out_edges()[(graph.csr_out_offsets()[u]+d)];
                    }
                }

                // printf("%ld %ld %ld %ld %ld %ld\n", thread_id,my_start,my_end, my_edges,graph.csr_out_offsets()[my_start],graph.csr_out_offsets()[my_end]);
                fflush(stdout);

                for(int64_t v=0; v<(nV+1); v++){
                    localOffset[v]=0;
                }
                for(int64_t v=(my_start); v<nV; v++){
                    localOffset[v+1] = localOffset[v] + (graph.csr_out_offsets()[v+1]-graph.csr_out_offsets()[v]);
                }

                HornetInit hornet_init((vert_t)nV, (vert_t)my_edges, localOffset, edges);
                HornetGraph hornet_graph(hornet_init);

                butterfly bfs(hornet_graph,fanout);

                #pragma omp barrier
                if(thread_id==0){
                    // TM.start();   
                    hipEventRecord(start); 
                    hipEventSynchronize(start); 
                }

                bfs.reset();    
                bfs.setInitValues(root, my_start, my_end,thread_id);

                bfs.queueRoot();

                #pragma omp barrier

                int front = 1;
                degree_t countTraversed=1;
                while(true){
                    bfs.oneIterationScan(front,isLrb);
                    bfComm[thread_id].queue_remote_ptr = bfs.remoteQueuePtr();
                    bfComm[thread_id].queue_remote_length = bfs.remoteQueueSize();

                    #pragma omp barrier

                    if(fanout==1){
                        for (int l=0; l<logNumGPUs; l++){
                            bfs.communication(bfComm,numGPUs,l);
         
                            bfComm[thread_id].queue_remote_length = bfs.remoteQueueSize();
                            #pragma omp barrier
                        }
                    }else if (fanout==4){
                        // if(numGPUs==4){
                        //     bfs.communication(bfComm,numGPUs,0);
         
                        //     bfComm[thread_id].queue_remote_length = bfs.remoteQueueSize();
                        //     #pragma omp barrier                        
                        // }
                        // else{ //if(numGPUs==16){

                            bfs.communication(bfComm,numGPUs,0);
                            bfComm[thread_id].queue_remote_length = bfs.remoteQueueSize();
                            #pragma omp barrier                        
                            if(numGPUs>4){
                                bfs.communication(bfComm,numGPUs,1);
                                bfComm[thread_id].queue_remote_length = bfs.remoteQueueSize();
                                #pragma omp barrier                                                        
                            }
                        // }
                    }
        // /            #pragma omp barrier

                    bfComm[thread_id].queue_remote_length = bfs.remoteQueueSize();
                    bfs.oneIterationComplete();

                    #pragma omp barrier
                    bfComm[thread_id].queue_local_length = bfs.localQueueSize();

                    #pragma omp barrier

                    degree_t currFrontier=0;
                    for(int64_t t=0; t<numGPUs; t++){
                        currFrontier+=bfComm[t].queue_local_length;
                        countTraversed+=bfComm[t].queue_local_length;
                    }

                    front++;
                    if(currFrontier==0){
                        break;
                    }
           
                }
                #pragma omp barrier

                if(thread_id==0){

                    // TM.stop();
                    // hipProfilerStop();
                    // TM.print("Butterfly BFS");
                    hipEventRecord(stop);
                    hipEventSynchronize(stop);
                    float milliseconds = 0;
                    hipEventElapsedTime(&milliseconds, start, stop);  
                    printf("%f,", milliseconds/1000.0);             
                    // std::cout << "Number of levels is : " << front << std::endl;
                    // std::cout << "The number of traversed vertices is : " << countTraversed << std::endl;
                }

                if(localOffset!=NULL)
                    free(localOffset);
                if(edges!=NULL)
                    free(edges);
            }
        }
        printf("\n");
    }

    }
    }


    omp_set_num_threads(maxGPUs);

    #pragma omp parallel
    {      
        int64_t thread_id = omp_get_thread_num ();
        hipSetDevice(thread_id);

        for(int64_t g=0; g<numGPUs; g++){
            if(g!=thread_id){
                int isCapable;
                hipDeviceCanAccessPeer(&isCapable,thread_id,g);
                if(isCapable==1){
                    hipDeviceDisablePeerAccess(g);
                }
            }
        }
    }

        return 0;
}
