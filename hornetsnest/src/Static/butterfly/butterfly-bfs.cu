#include "hip/hip_runtime.h"



#include "Static/butterfly/butterfly-bfs.cuh"
#include "Static/butterfly/butterfly-bfsOperators.cuh"


using length_t = int;
using namespace std;
using namespace hornets_nest::gpu;
namespace hornets_nest {


struct countDegrees {
    int32_t *bins; 


    OPERATOR(Vertex& vertex) {

        __shared__ int32_t localBins[33];
        int id = threadIdx.x;
        if(id==0){
            for (int i=0; i<33; i++)
            localBins[i]=0;
        }
        __syncthreads();

        int32_t size = vertex.degree();
        int32_t myBin  = __clz(size);

        int32_t my_pos = atomicAdd(localBins+myBin, 1);

        __syncthreads();

       if(id==0){
            for (int i=0; i<33; i++){            
                atomicAdd(bins+i, localBins[i]);
            }

        }

    }
};





__global__ void  binPrefixKernel(int32_t     *bins, int32_t     *d_binsPrefix){

    int i = threadIdx.x + blockIdx.x *blockDim.x;
    if(i>=1)
        return;
    d_binsPrefix[0]=0;
    for(int b=0; b<33; b++){
        d_binsPrefix[b+1]=d_binsPrefix[b]+bins[b];
        // printf("%d ",d_binsPrefix[b+1] );  
    }
    // printf("\n");


}

template<typename HornetDevice>
__global__ void  rebinKernel(
  HornetDevice hornet ,
  const vert_t    *original,
  int32_t    *d_binsPrefix,
  vert_t     *d_reOrg,
  int N){

    int i = threadIdx.x + blockIdx.x *blockDim.x;

    __shared__ int32_t localBins[33];
    __shared__ int32_t localPos[33];

    __shared__ int32_t prefix[33];    
    int id = threadIdx.x;
    if(id<33){
      localBins[id]=0;
      localPos[id]=0;
    }

    __syncthreads();

    int myBin,myPos;
    if(i<N){
        int32_t adjSize= hornet.vertex(original[i]).degree();
        myBin  = __clz(adjSize);
        myPos = atomicAdd(localBins+myBin, 1);
    }


  __syncthreads();
    if(id<33){
        localPos[id]=atomicAdd(d_binsPrefix+id, localBins[id]);
    }
  __syncthreads();

    if(i<N){
        int pos = localPos[myBin]+myPos;
        d_reOrg[pos]=original[i];
    }

}









/// TODO - changed hostKatzdata to pointer so that I can try to inherit it in
// the streaming case.

butterfly::butterfly(HornetGraph& hornet, int fanout_) :
                                       StaticAlgorithm(hornet),
                                       load_balancing(hornet)
{

    fanout=fanout_;
    if(fanout!=4 && fanout!=1){
        printf("Fanout has to be 1 or 4 for butterfly `BFS\n");
        exit(0);
    }

    hd_bfsData().currLevel=0;

    gpu::allocate(hd_bfsData().d_buffer, fanout*hornet.nV());
    gpu::allocate(hd_bfsData().d_Marked, hornet.nV());
    gpu::allocate(hd_bfsData().d_dist, hornet.nV());

    gpu::allocate(hd_bfsData().d_lrbRelabled, hornet.nV());
    gpu::allocate(hd_bfsData().d_bins, 33);
    gpu::allocate(hd_bfsData().d_binsPrefix, 33);


    // gpu::allocate(hd_bfsData().queueRemote, hornet.nV());
    // hd_bfsData().queueRemoteSize=0;

    // hd_bfsData().queueLocal.initialize(hornet);
    // hd_bfsData().queueRemote.initialize(hornet);

    hd_bfsData().queueLocal.initialize((size_t)hornet.nV());
    hd_bfsData().queueRemote.initialize((size_t)hornet.nV());


    reset();
}

butterfly::~butterfly() {
    release();
}

void butterfly::setInitValues(vert_t root_ ,vert_t lower_, vert_t upper_,int64_t gpu_id_)
{
    // if(gpu_id_==0)
    //     std::cout << " " << gpu_id_ << " " << lower_ << " " <<  upper_ << " " <<  root_ << std::endl;
    // hd_bfsData.sync();

    hd_bfsData().currLevel  = 1;
    hd_bfsData().root       = root_;
    hd_bfsData().lower      = lower_;
    hd_bfsData().upper      = upper_;
    hd_bfsData().gpu_id     = gpu_id_;
// /    hd_bfsData().queueRemoteSize=0;

    // if(gpu_id_==0)
    //     std::cout << " " << hd_bfsData().gpu_id << " " << hd_bfsData().lower << " " <<  hd_bfsData().upper << " " <<  hd_bfsData().root << std::endl;

    // if(gpu_id_==0)
    //     std::cout << " " << hd_bfsData().gpu_id << " " << hd_bfsData().lower << " " <<  hd_bfsData().upper << " " <<  hd_bfsData().root << std::endl;

}


void butterfly::reset() {

    forAllnumV(hornet, InitBFS { hd_bfsData });
    hipDeviceSynchronize();

    // hd_bfsData.sync();
}

void butterfly::release(){
    gpu::free(hd_bfsData().d_buffer);
    gpu::free(hd_bfsData().d_Marked);
    gpu::free(hd_bfsData().d_dist);
    gpu::free(hd_bfsData().d_lrbRelabled);
    gpu::free(hd_bfsData().d_bins);
    gpu::free(hd_bfsData().d_binsPrefix);


    // gpu::free(hd_bfsData().queueRemote);

}

void butterfly::queueRoot(){
    
    // std::cout << " " << hd_bfsData().gpu_id << " " << hd_bfsData().lower << " " <<  hd_bfsData().upper << " " <<  hd_bfsData().root << std::endl;


    if (hd_bfsData().root >= hd_bfsData().lower && hd_bfsData().root <hd_bfsData().upper){
        hd_bfsData().queueLocal.insert(hd_bfsData().root);                   // insert source in the frontier
    }
    gpu::memsetZero(hd_bfsData().d_dist + hd_bfsData().root);

}


void butterfly::oneIterationScan(degree_t level,bool lrb){

    hd_bfsData().currLevel = level;
    if (hd_bfsData().queueLocal.size() > 0) {
        if(!lrb){
            forAllEdges(hornet, hd_bfsData().queueLocal, BFSTopDown_One_Iter { hd_bfsData },load_balancing);
        }
        else{
            // hd_bfsData().queueLocal
            int32_t elements = hd_bfsData().queueLocal.size();

            gpu::memsetZero(hd_bfsData().d_bins, 33);            

            forAllVertices(hornet, hd_bfsData().queueLocal,countDegrees{hd_bfsData().d_bins});

            binPrefixKernel <<<1,32>>> (hd_bfsData().d_bins,hd_bfsData().d_binsPrefix);  

            int32_t h_binsPrefix[33];
            hipMemcpy(h_binsPrefix, hd_bfsData().d_binsPrefix,sizeof(int32_t)*33, hipMemcpyDeviceToHost);

            // for(int i=0; i<33; i++){
            //     printf("%d ",h_binsPrefix[i]);
            // }
            // printf("\n" );

            const int RB_BLOCK_SIZE = 256;
            int rebinblocks = (elements)/RB_BLOCK_SIZE + (((elements)%RB_BLOCK_SIZE)?1:0);

            if(rebinblocks){
              rebinKernel<<<rebinblocks,RB_BLOCK_SIZE>>>(hornet.device(),hd_bfsData().queueLocal.device_input_ptr(),
                hd_bfsData().d_binsPrefix, hd_bfsData().d_lrbRelabled,elements);
            }


            // if(rebinblocks>0)
            //     BFSTopDown_One_Iter_kernel<<<rebinblocks,RB_BLOCK_SIZE>>>(hornet.device_side(),
            //         hd_bfsData,elements,0);

            const int bi = 26;
            // printf("starting point is %d\n",h_binsPrefix[bi]);
            // hipStream_t streams[2];
            //   hipStreamCreate ( &(streams[0]));
            //   hipStreamCreate ( &(streams[1]));


            rebinblocks = (h_binsPrefix[bi]);
            if(rebinblocks>0){
                // printf("fat is running %d \n",h_binsPrefix[bi]);
                BFSTopDown_One_Iter_kernel_fat<<<rebinblocks,RB_BLOCK_SIZE>>>(hornet.device(),hd_bfsData,h_binsPrefix[bi]);            }



            rebinblocks = (elements-h_binsPrefix[bi])/RB_BLOCK_SIZE + (((elements-h_binsPrefix[bi])%RB_BLOCK_SIZE)?1:0);
            if(rebinblocks>0)
                BFSTopDown_One_Iter_kernel<<<rebinblocks,RB_BLOCK_SIZE>>>(hornet.device(),
                    hd_bfsData,elements-h_binsPrefix[bi],h_binsPrefix[bi]);


        }

        // hd_bfsData.sync();

        // std::cout << hd_bfsData().gpu_id << " " << hd_bfsData().queueLocal.size_sync_out() << std::endl;
        // std::cout << hd_bfsData().gpu_id << " " << hd_bfsData().queueRemote.size_sync_out() << std::endl;
        // hd_bfsData().queueLocal.swap();
        // std::cout << hd_bfsData().gpu_id << " " << hd_bfsData().queueLocal.size() << std::endl;

    }

}

void butterfly::oneIterationComplete(){

    hd_bfsData().queueLocal.swap();

    hd_bfsData().queueRemote.clear();
    hipDeviceSynchronize();

}


void butterfly::communication(butterfly_communication* bfComm, int numGPUs, int iteration){

    if(fanout==1){

        int but_net[4][16] = {
                                {1,0,3,2,5,4,7,6,9,8,11,10,13,12,15,14},
                                {2,3,0,1,6,7,4,5,10,11,8,9,14,15,12,13},
                                {4,5,6,7,0,1,2,3,12,13,14,15,8,9,10,11},
                                {8,9,10,11,12,13,14,15,0,1,2,3,4,5,6,7}
        };  

        int my_gpu = hd_bfsData().gpu_id;
        int copy_gpu=but_net[iteration][my_gpu];
        if(copy_gpu>=numGPUs){
            copy_gpu=numGPUs-1;
        }

        hd_bfsData().h_bufferSize=bfComm[copy_gpu].queue_remote_length;
        hipMemcpyPeerAsync(hd_bfsData().d_buffer, my_gpu, bfComm[copy_gpu].queue_remote_ptr,copy_gpu, hd_bfsData().h_bufferSize*sizeof(vert_t));
        
        if (hd_bfsData().h_bufferSize > 0){
            forAllVertices(hornet, hd_bfsData().d_buffer, hd_bfsData().h_bufferSize, NeighborUpdates { hd_bfsData });

        }

    }else if(fanout==4){
        int but_net_first[16][4]={{0,1,2,3},{0,1,2,3},{0,1,2,3},{0,1,2,3},
                                  {4,5,6,7},{4,5,6,7},{4,5,6,7},{4,5,6,7},
                                  {8,9,10,11},{8,9,10,11},{8,9,10,11},{8,9,10,11},
                                  {12,13,14,15},{12,13,14,15},{12,13,14,15},{12,13,14,15}};

        int but_net_second[16][4]={{0,4,8,12},{1,5,9,13},{2,6,10,14},{3,7,11,15},
                                  {0,4,8,12},{1,5,9,13},{2,6,10,14},{3,7,11,15},
                                  {0,4,8,12},{1,5,9,13},{2,6,10,14},{3,7,11,15},
                                  {0,4,8,12},{1,5,9,13},{2,6,10,14},{3,7,11,15}};

        int my_gpu = hd_bfsData().gpu_id;

        hd_bfsData().h_bufferSize=0;
        int pos=0;
        for(int s=0; s<4;s++){
            int copy_gpu;
            if(iteration==0)
                copy_gpu=but_net_first[my_gpu][s];
            else
                copy_gpu=but_net_second[my_gpu][s];

            if(copy_gpu>=numGPUs){
                copy_gpu=numGPUs-1;
            }

            
            if(my_gpu!=copy_gpu){
                int remoteLength = bfComm[copy_gpu].queue_remote_length;                
                hipMemcpyPeerAsync(hd_bfsData().d_buffer+pos, my_gpu, bfComm[copy_gpu].queue_remote_ptr,copy_gpu, remoteLength*sizeof(vert_t));
                pos+=remoteLength;
                hd_bfsData().h_bufferSize+=remoteLength;

            }
        }
        
        if (hd_bfsData().h_bufferSize > 0){
            forAllVertices(hornet, hd_bfsData().d_buffer, hd_bfsData().h_bufferSize, NeighborUpdates { hd_bfsData });

        }


    }


}



void butterfly::run() {



}


// int butterfly::getDepth() {
//     return hd_bfsData();
// }

bool butterfly::validate() {
    return true;
}

} // namespace hornets_nest
